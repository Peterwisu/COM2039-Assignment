/*
 ============================================================================
 Name        : Q2e.cu
 Author      : Wish Suharitdamrong
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
/*************************************
**Question 2e
*************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 *
 * Initialize value in global memory
 *
 */
void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
    printf("%d ,%d\n",i,a[i]);


  }
}

/**
 *
 * Kernel function to double a value in each index
 *
 */
__global__ void doubleElements(int *a, int N)
{
  int i;
  // calculate the index of the thread
  i = blockIdx.x * blockDim.x + threadIdx.x;
  // if the index is less than the size of the array
  if (i < N)
  {

	// multiply the value of the array at index i by 2
    a[i] *= 2;
    // print out to check the value assigned in the array
    printf("%d ,%d\n",i,a[i]);



  }
}


/**
 *
 *  Check the value in each index is double value of its index
 *
 */
bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
	printf("%d ,%d\n",i,a[i]);
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  //  number of thread
  int N = 100;

  int *a;
  // size of memory
  size_t size = N * sizeof(int);
  // Allocate memory into Unified memory which is accessible for both GPU and CPU
  hipMallocManaged(&a,size);

  // call init function to initialise a value of array n
  init(a, N);
  // initialise number of blocks and thread for Gridsize and BLocksize
  size_t threads_per_block = 10;
  size_t number_of_blocks = 10;


  // Call Kernel
  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  // wait for gpu to finish execute
  hipDeviceSynchronize();


  // Call function to check value is double
  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  // Free unified memory
  hipFree(a);

}


