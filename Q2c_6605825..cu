/*
 ============================================================================
 Name        : Q2c.cu
 Author      : Wish Suharitdamrong
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
/*************************************
**Question 2c
*************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop()
{

    printf("This is iteration number %d\n", threadIdx.x);


}

int main()
{
  /*
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this question, only use 1 block of threads.
   */

  // Declare Number of threads
  int N = 10;
  // Number of threads is a block since the question specify only use 1 block of thread
  int blockSize = N;
  // Declare number of block since the question specify only use 1 block of thread
  int block_num =1;
  // Call Kernel
  loop<<<block_num,blockSize>>>();
  // Wait for GPU to finish execute
  hipDeviceSynchronize();
  return 0;
}
