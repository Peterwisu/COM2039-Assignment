/*
 ============================================================================
 Name        : Q2d.cu
 Author      : Wish Suharitdamrong
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
/*************************************
**Question 2c
*************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop(int N)
{
	// calculate the index of the thread
	int tid = threadIdx.x+blockDim.x*blockIdx.x;
	// only print if the index is less than the size of the array N
	if(tid<N){
		printf("This is iteration number %d\n",tid);
	}
}

int main(void)
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this question, use at leat 2 blocks of threads.
   */

	// Declare Number of thread in one block
	float N = 10;
	// Decalre Number of thread in block
	int blockSize = 3;
	// Declare Number of block
	int block_num =ceil(N/blockSize);
	// check the number of block size
	printf("%d \n",block_num);
	// Call kernel
	loop<<<block_num,blockSize>>>(N);
	// Wait for GPU to finish execute
	hipDeviceSynchronize();

	return 0;
}
