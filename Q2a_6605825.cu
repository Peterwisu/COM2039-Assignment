/*
 ============================================================================
 Name        : Q2a.cu
 Author      : Wish Suharitdamrong
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

/*************************************
**Question 2a
*************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * Function to print Hello from the CPU
 *
 */
void helloCPU(){
    printf("Hello from the CPU.\n");
}
/**
 * Function use a kernel to print Hello from GPU
 *
 */
__global__ void helloGPU()
{
  printf("Hello  from the GPU.\n");
}

int main(void)
{

  // Call  Kernel to print Hello from GPU with 1 block and 1 thread so that it's execute only 1 times.
  helloGPU<<<1,1>>>();
  // Wait for GPU to finish execute
  hipDeviceSynchronize();
  // print Hello from the CPU
  helloCPU();
  return 0;
}
