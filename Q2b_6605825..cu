/*
 ============================================================================
 Name        : Q2b.cu
 Author      : Wish Suharitdamrong
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
/*************************************
**Question 2b
*************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printSuccessForCorrectExecutionConfiguration()
{

  /**
   *
   * Print 'Success' only when the thread Id is 1023 and block Id is 255
   *
   */
  if(threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("Success!\n threadIDx : % d , BlockIdx : %d\n",threadIdx.x,blockIdx.x);

  }

}

int main(void)
{
  /*
   * Update the execution configuration so that the kernel
   * will print `"Success!"`.
   */



	// To print the success from the kernel we need the kernel to run thread id 1023 at the block id 255
	// Declare Number of block to 1024 (0-1023)
	int blockSize =1024;
	// Declare Number of threads in a block (0-255)
	int grid_size = 256;
	// Call Kernel
	printSuccessForCorrectExecutionConfiguration<<<grid_size, blockSize>>>();
	// Wait for GPU to finish execute
	hipDeviceSynchronize();

	return 0;

}
